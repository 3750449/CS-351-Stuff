
#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <string>
#include <vector>

using Count = size_t;
using DataType = long;

const DataType DefaultStartValue = -6.0;
const Count TestSize = 1'000'000'000;
const Count NumCheckValues = 500;

//
// CUDA kernel to perform iota: fill array with startValue + i
//
__global__ void iota(size_t n, DataType* x, DataType startValue) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;
    x[i] = startValue + static_cast<DataType>(i);
}

int main(int argc, char* argv[]) {
    Count numValues = argc > 1 ? std::stol(argv[1]) : TestSize;

    std::vector<DataType> values(numValues);
    size_t numBytes = numValues * sizeof(DataType);

    DataType* gpuValues;
    hipMalloc(&gpuValues, numBytes);
    hipMemcpy(gpuValues, values.data(), numBytes, hipMemcpyHostToDevice);

    DataType startValue = DefaultStartValue;

    int chunkSize = 256;
    int numChunks = (numValues + chunkSize - 1) / chunkSize;
    iota<<<numChunks, chunkSize>>>(numValues, gpuValues, startValue);

    hipMemcpy(values.data(), gpuValues, numBytes, hipMemcpyDeviceToHost);

    Count step = numValues / NumCheckValues;
    for (int i = 6, n = 0; i < numValues && n < NumCheckValues; ++n, i += step) {
        DataType checkValue = startValue + static_cast<DataType>(i);

        if (values[i] != checkValue) {
            std::cerr << "Values do not match for position " << i
                      << ": " << values[i] << " != " << checkValue << "\n";
            exit(EXIT_FAILURE);
        }
    }

    std::cout << "All values validated successfully.\n";
    hipFree(gpuValues);
    return 0;
}

